#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <ctime>
#define BLOCKS				1024
#define THREADS				1024
#define TRIALS				1024*1024

// __byte_perm Constants
#define SHIFT_1_RIGHT			17185  // 0x00004321U i.e. ( >> 8 )
#define SHIFT_2_RIGHT			21554  // 0x00005432U i.e. ( >> 16 )
#define SHIFT_3_RIGHT			25923  // 0x00006543U i.e. ( >> 24 )

__int64 trial = 1, keys = 10;
double PCFreq = 0.0;
__int64 CounterStart = 0;
#define bit8 unsigned char
#define bit32 unsigned int
#define bit64 unsigned __int64 

bit32 T0[256] = { 0xee777799, 0xe874749c, 0xf47a7a8e, 0xf279798b, 0xe2717193, 0xfe7f7f81, 0xf67b7b8d, 0xe0707090, 0xf87c7c84, 0xe6737395, 0xe4727296, 0xec76769a, 0xf0787888, 0xfc7e7e82, 0xfa7d7d87, 0xea75759f, 0x8e4747c9, 0x884444cc, 0x944a4ade, 0x924949db, 0x824141c3, 0x9e4f4fd1, 0x964b4bdd, 0x804040c0, 0x984c4cd4, 0x864343c5, 0x844242c6, 0x8c4646ca, 0x904848d8, 0x9c4e4ed2, 0x9a4d4dd7, 0x8a4545cf, 0x55a7a7f2, 0x53a4a4f7, 0x4faaaae5, 0x49a9a9e0, 0x59a1a1f8, 0x45afafea, 0x4dababe6, 0x5ba0a0fb, 0x43acacef, 0x5da3a3fe, 0x5fa2a2fd, 0x57a6a6f1, 0x4ba8a8e3, 0x47aeaee9, 0x41adadec, 0x51a5a5f4, 0x359797a2, 0x339494a7, 0x2f9a9ab5, 0x299999b0, 0x399191a8, 0x259f9fba, 0x2d9b9bb6, 0x3b9090ab, 0x239c9cbf, 0x3d9393ae, 0x3f9292ad, 0x379696a1, 0x2b9898b3, 0x279e9eb9, 0x219d9dbc, 0x319595a4, 0x2e171739, 0x2814143c, 0x341a1a2e, 0x3219192b, 0x22111133, 0x3e1f1f21, 0x361b1b2d, 0x20101030, 0x381c1c24, 0x26131335, 0x24121236, 0x2c16163a, 0x30181828, 0x3c1e1e22, 0x3a1d1d27, 0x2a15153f, 0xf5f7f702, 0xf3f4f407, 0xeffafa15, 0xe9f9f910, 0xf9f1f108, 0xe5ffff1a, 0xedfbfb16, 0xfbf0f00b, 0xe3fcfc1f, 0xfdf3f30e, 0xfff2f20d, 0xf7f6f601, 0xebf8f813, 0xe7fefe19, 0xe1fdfd1c, 0xf1f5f504, 0x75b7b7c2, 0x73b4b4c7, 0x6fbabad5, 0x69b9b9d0, 0x79b1b1c8, 0x65bfbfda, 0x6dbbbbd6, 0x7bb0b0cb, 0x63bcbcdf, 0x7db3b3ce, 0x7fb2b2cd, 0x77b6b6c1, 0x6bb8b8d3, 0x67bebed9, 0x61bdbddc, 0x71b5b5c4, 0x0e070709, 0x0804040c, 0x140a0a1e, 0x1209091b, 0x02010103, 0x1e0f0f11, 0x160b0b1d, 0x00000000, 0x180c0c14, 0x06030305, 0x04020206, 0x0c06060a, 0x10080818, 0x1c0e0e12, 0x1a0d0d17, 0x0a05050f, 0x95c7c752, 0x93c4c457, 0x8fcaca45, 0x89c9c940, 0x99c1c158, 0x85cfcf4a, 0x8dcbcb46, 0x9bc0c05b, 0x83cccc4f, 0x9dc3c35e, 0x9fc2c25d, 0x97c6c651, 0x8bc8c843, 0x87cece49, 0x81cdcd4c, 0x91c5c554, 0x6e373759, 0x6834345c, 0x743a3a4e, 0x7239394b, 0x62313153, 0x7e3f3f41, 0x763b3b4d, 0x60303050, 0x783c3c44, 0x66333355, 0x64323256, 0x6c36365a, 0x70383848, 0x7c3e3e42, 0x7a3d3d47, 0x6a35355f, 0x4e272769, 0x4824246c, 0x542a2a7e, 0x5229297b, 0x42212163, 0x5e2f2f71, 0x562b2b7d, 0x40202060, 0x582c2c74, 0x46232365, 0x44222266, 0x4c26266a, 0x50282878, 0x5c2e2e72, 0x5a2d2d77, 0x4a25256f, 0xce6767a9, 0xc86464ac, 0xd46a6abe, 0xd26969bb, 0xc26161a3, 0xde6f6fb1, 0xd66b6bbd, 0xc06060a0, 0xd86c6cb4, 0xc66363a5, 0xc46262a6, 0xcc6666aa, 0xd06868b8, 0xdc6e6eb2, 0xda6d6db7, 0xca6565af, 0x15878792, 0x13848497, 0x0f8a8a85, 0x09898980, 0x19818198, 0x058f8f8a, 0x0d8b8b86, 0x1b80809b, 0x038c8c8f, 0x1d83839e, 0x1f82829d, 0x17868691, 0x0b888883, 0x078e8e89, 0x018d8d8c, 0x11858594, 0xd5e7e732, 0xd3e4e437, 0xcfeaea25, 0xc9e9e920, 0xd9e1e138, 0xc5efef2a, 0xcdebeb26, 0xdbe0e03b, 0xc3ecec2f, 0xdde3e33e, 0xdfe2e23d, 0xd7e6e631, 0xcbe8e823, 0xc7eeee29, 0xc1eded2c, 0xd1e5e534, 0xb5d7d762, 0xb3d4d467, 0xafdada75, 0xa9d9d970, 0xb9d1d168, 0xa5dfdf7a, 0xaddbdb76, 0xbbd0d06b, 0xa3dcdc7f, 0xbdd3d36e, 0xbfd2d26d, 0xb7d6d661, 0xabd8d873, 0xa7dede79, 0xa1dddd7c, 0xb1d5d564, 0xae5757f9, 0xa85454fc, 0xb45a5aee, 0xb25959eb, 0xa25151f3, 0xbe5f5fe1, 0xb65b5bed, 0xa05050f0, 0xb85c5ce4, 0xa65353f5, 0xa45252f6, 0xac5656fa, 0xb05858e8, 0xbc5e5ee2, 0xba5d5de7, 0xaa5555ff};
bit32 T1[256] = { 0x99ee7777, 0x9ce87474, 0x8ef47a7a, 0x8bf27979, 0x93e27171, 0x81fe7f7f, 0x8df67b7b, 0x90e07070, 0x84f87c7c, 0x95e67373, 0x96e47272, 0x9aec7676, 0x88f07878, 0x82fc7e7e, 0x87fa7d7d, 0x9fea7575, 0xc98e4747, 0xcc884444, 0xde944a4a, 0xdb924949, 0xc3824141, 0xd19e4f4f, 0xdd964b4b, 0xc0804040, 0xd4984c4c, 0xc5864343, 0xc6844242, 0xca8c4646, 0xd8904848, 0xd29c4e4e, 0xd79a4d4d, 0xcf8a4545, 0xf255a7a7, 0xf753a4a4, 0xe54faaaa, 0xe049a9a9, 0xf859a1a1, 0xea45afaf, 0xe64dabab, 0xfb5ba0a0, 0xef43acac, 0xfe5da3a3, 0xfd5fa2a2, 0xf157a6a6, 0xe34ba8a8, 0xe947aeae, 0xec41adad, 0xf451a5a5, 0xa2359797, 0xa7339494, 0xb52f9a9a, 0xb0299999, 0xa8399191, 0xba259f9f, 0xb62d9b9b, 0xab3b9090, 0xbf239c9c, 0xae3d9393, 0xad3f9292, 0xa1379696, 0xb32b9898, 0xb9279e9e, 0xbc219d9d, 0xa4319595, 0x392e1717, 0x3c281414, 0x2e341a1a, 0x2b321919, 0x33221111, 0x213e1f1f, 0x2d361b1b, 0x30201010, 0x24381c1c, 0x35261313, 0x36241212, 0x3a2c1616, 0x28301818, 0x223c1e1e, 0x273a1d1d, 0x3f2a1515, 0x02f5f7f7, 0x07f3f4f4, 0x15effafa, 0x10e9f9f9, 0x08f9f1f1, 0x1ae5ffff, 0x16edfbfb, 0x0bfbf0f0, 0x1fe3fcfc, 0x0efdf3f3, 0x0dfff2f2, 0x01f7f6f6, 0x13ebf8f8, 0x19e7fefe, 0x1ce1fdfd, 0x04f1f5f5, 0xc275b7b7, 0xc773b4b4, 0xd56fbaba, 0xd069b9b9, 0xc879b1b1, 0xda65bfbf, 0xd66dbbbb, 0xcb7bb0b0, 0xdf63bcbc, 0xce7db3b3, 0xcd7fb2b2, 0xc177b6b6, 0xd36bb8b8, 0xd967bebe, 0xdc61bdbd, 0xc471b5b5, 0x090e0707, 0x0c080404, 0x1e140a0a, 0x1b120909, 0x03020101, 0x111e0f0f, 0x1d160b0b, 0x00000000, 0x14180c0c, 0x05060303, 0x06040202, 0x0a0c0606, 0x18100808, 0x121c0e0e, 0x171a0d0d, 0x0f0a0505, 0x5295c7c7, 0x5793c4c4, 0x458fcaca, 0x4089c9c9, 0x5899c1c1, 0x4a85cfcf, 0x468dcbcb, 0x5b9bc0c0, 0x4f83cccc, 0x5e9dc3c3, 0x5d9fc2c2, 0x5197c6c6, 0x438bc8c8, 0x4987cece, 0x4c81cdcd, 0x5491c5c5, 0x596e3737, 0x5c683434, 0x4e743a3a, 0x4b723939, 0x53623131, 0x417e3f3f, 0x4d763b3b, 0x50603030, 0x44783c3c, 0x55663333, 0x56643232, 0x5a6c3636, 0x48703838, 0x427c3e3e, 0x477a3d3d, 0x5f6a3535, 0x694e2727, 0x6c482424, 0x7e542a2a, 0x7b522929, 0x63422121, 0x715e2f2f, 0x7d562b2b, 0x60402020, 0x74582c2c, 0x65462323, 0x66442222, 0x6a4c2626, 0x78502828, 0x725c2e2e, 0x775a2d2d, 0x6f4a2525, 0xa9ce6767, 0xacc86464, 0xbed46a6a, 0xbbd26969, 0xa3c26161, 0xb1de6f6f, 0xbdd66b6b, 0xa0c06060, 0xb4d86c6c, 0xa5c66363, 0xa6c46262, 0xaacc6666, 0xb8d06868, 0xb2dc6e6e, 0xb7da6d6d, 0xafca6565, 0x92158787, 0x97138484, 0x850f8a8a, 0x80098989, 0x98198181, 0x8a058f8f, 0x860d8b8b, 0x9b1b8080, 0x8f038c8c, 0x9e1d8383, 0x9d1f8282, 0x91178686, 0x830b8888, 0x89078e8e, 0x8c018d8d, 0x94118585, 0x32d5e7e7, 0x37d3e4e4, 0x25cfeaea, 0x20c9e9e9, 0x38d9e1e1, 0x2ac5efef, 0x26cdebeb, 0x3bdbe0e0, 0x2fc3ecec, 0x3edde3e3, 0x3ddfe2e2, 0x31d7e6e6, 0x23cbe8e8, 0x29c7eeee, 0x2cc1eded, 0x34d1e5e5, 0x62b5d7d7, 0x67b3d4d4, 0x75afdada, 0x70a9d9d9, 0x68b9d1d1, 0x7aa5dfdf, 0x76addbdb, 0x6bbbd0d0, 0x7fa3dcdc, 0x6ebdd3d3, 0x6dbfd2d2, 0x61b7d6d6, 0x73abd8d8, 0x79a7dede, 0x7ca1dddd, 0x64b1d5d5, 0xf9ae5757, 0xfca85454, 0xeeb45a5a, 0xebb25959, 0xf3a25151, 0xe1be5f5f, 0xedb65b5b, 0xf0a05050, 0xe4b85c5c, 0xf5a65353, 0xf6a45252, 0xfaac5656, 0xe8b05858, 0xe2bc5e5e, 0xe7ba5d5d, 0xffaa5555};
bit32 T2[256] = { 0x7799ee77, 0x749ce874, 0x7a8ef47a, 0x798bf279, 0x7193e271, 0x7f81fe7f, 0x7b8df67b, 0x7090e070, 0x7c84f87c, 0x7395e673, 0x7296e472, 0x769aec76, 0x7888f078, 0x7e82fc7e, 0x7d87fa7d, 0x759fea75, 0x47c98e47, 0x44cc8844, 0x4ade944a, 0x49db9249, 0x41c38241, 0x4fd19e4f, 0x4bdd964b, 0x40c08040, 0x4cd4984c, 0x43c58643, 0x42c68442, 0x46ca8c46, 0x48d89048, 0x4ed29c4e, 0x4dd79a4d, 0x45cf8a45, 0xa7f255a7, 0xa4f753a4, 0xaae54faa, 0xa9e049a9, 0xa1f859a1, 0xafea45af, 0xabe64dab, 0xa0fb5ba0, 0xacef43ac, 0xa3fe5da3, 0xa2fd5fa2, 0xa6f157a6, 0xa8e34ba8, 0xaee947ae, 0xadec41ad, 0xa5f451a5, 0x97a23597, 0x94a73394, 0x9ab52f9a, 0x99b02999, 0x91a83991, 0x9fba259f, 0x9bb62d9b, 0x90ab3b90, 0x9cbf239c, 0x93ae3d93, 0x92ad3f92, 0x96a13796, 0x98b32b98, 0x9eb9279e, 0x9dbc219d, 0x95a43195, 0x17392e17, 0x143c2814, 0x1a2e341a, 0x192b3219, 0x11332211, 0x1f213e1f, 0x1b2d361b, 0x10302010, 0x1c24381c, 0x13352613, 0x12362412, 0x163a2c16, 0x18283018, 0x1e223c1e, 0x1d273a1d, 0x153f2a15, 0xf702f5f7, 0xf407f3f4, 0xfa15effa, 0xf910e9f9, 0xf108f9f1, 0xff1ae5ff, 0xfb16edfb, 0xf00bfbf0, 0xfc1fe3fc, 0xf30efdf3, 0xf20dfff2, 0xf601f7f6, 0xf813ebf8, 0xfe19e7fe, 0xfd1ce1fd, 0xf504f1f5, 0xb7c275b7, 0xb4c773b4, 0xbad56fba, 0xb9d069b9, 0xb1c879b1, 0xbfda65bf, 0xbbd66dbb, 0xb0cb7bb0, 0xbcdf63bc, 0xb3ce7db3, 0xb2cd7fb2, 0xb6c177b6, 0xb8d36bb8, 0xbed967be, 0xbddc61bd, 0xb5c471b5, 0x07090e07, 0x040c0804, 0x0a1e140a, 0x091b1209, 0x01030201, 0x0f111e0f, 0x0b1d160b, 0x00000000, 0x0c14180c, 0x03050603, 0x02060402, 0x060a0c06, 0x08181008, 0x0e121c0e, 0x0d171a0d, 0x050f0a05, 0xc75295c7, 0xc45793c4, 0xca458fca, 0xc94089c9, 0xc15899c1, 0xcf4a85cf, 0xcb468dcb, 0xc05b9bc0, 0xcc4f83cc, 0xc35e9dc3, 0xc25d9fc2, 0xc65197c6, 0xc8438bc8, 0xce4987ce, 0xcd4c81cd, 0xc55491c5, 0x37596e37, 0x345c6834, 0x3a4e743a, 0x394b7239, 0x31536231, 0x3f417e3f, 0x3b4d763b, 0x30506030, 0x3c44783c, 0x33556633, 0x32566432, 0x365a6c36, 0x38487038, 0x3e427c3e, 0x3d477a3d, 0x355f6a35, 0x27694e27, 0x246c4824, 0x2a7e542a, 0x297b5229, 0x21634221, 0x2f715e2f, 0x2b7d562b, 0x20604020, 0x2c74582c, 0x23654623, 0x22664422, 0x266a4c26, 0x28785028, 0x2e725c2e, 0x2d775a2d, 0x256f4a25, 0x67a9ce67, 0x64acc864, 0x6abed46a, 0x69bbd269, 0x61a3c261, 0x6fb1de6f, 0x6bbdd66b, 0x60a0c060, 0x6cb4d86c, 0x63a5c663, 0x62a6c462, 0x66aacc66, 0x68b8d068, 0x6eb2dc6e, 0x6db7da6d, 0x65afca65, 0x87921587, 0x84971384, 0x8a850f8a, 0x89800989, 0x81981981, 0x8f8a058f, 0x8b860d8b, 0x809b1b80, 0x8c8f038c, 0x839e1d83, 0x829d1f82, 0x86911786, 0x88830b88, 0x8e89078e, 0x8d8c018d, 0x85941185, 0xe732d5e7, 0xe437d3e4, 0xea25cfea, 0xe920c9e9, 0xe138d9e1, 0xef2ac5ef, 0xeb26cdeb, 0xe03bdbe0, 0xec2fc3ec, 0xe33edde3, 0xe23ddfe2, 0xe631d7e6, 0xe823cbe8, 0xee29c7ee, 0xed2cc1ed, 0xe534d1e5, 0xd762b5d7, 0xd467b3d4, 0xda75afda, 0xd970a9d9, 0xd168b9d1, 0xdf7aa5df, 0xdb76addb, 0xd06bbbd0, 0xdc7fa3dc, 0xd36ebdd3, 0xd26dbfd2, 0xd661b7d6, 0xd873abd8, 0xde79a7de, 0xdd7ca1dd, 0xd564b1d5, 0x57f9ae57, 0x54fca854, 0x5aeeb45a, 0x59ebb259, 0x51f3a251, 0x5fe1be5f, 0x5bedb65b, 0x50f0a050, 0x5ce4b85c, 0x53f5a653, 0x52f6a452, 0x56faac56, 0x58e8b058, 0x5ee2bc5e, 0x5de7ba5d, 0x55ffaa55};
bit32 T3[256] = { 0x777799ee, 0x74749ce8, 0x7a7a8ef4, 0x79798bf2, 0x717193e2, 0x7f7f81fe, 0x7b7b8df6, 0x707090e0, 0x7c7c84f8, 0x737395e6, 0x727296e4, 0x76769aec, 0x787888f0, 0x7e7e82fc, 0x7d7d87fa, 0x75759fea, 0x4747c98e, 0x4444cc88, 0x4a4ade94, 0x4949db92, 0x4141c382, 0x4f4fd19e, 0x4b4bdd96, 0x4040c080, 0x4c4cd498, 0x4343c586, 0x4242c684, 0x4646ca8c, 0x4848d890, 0x4e4ed29c, 0x4d4dd79a, 0x4545cf8a, 0xa7a7f255, 0xa4a4f753, 0xaaaae54f, 0xa9a9e049, 0xa1a1f859, 0xafafea45, 0xababe64d, 0xa0a0fb5b, 0xacacef43, 0xa3a3fe5d, 0xa2a2fd5f, 0xa6a6f157, 0xa8a8e34b, 0xaeaee947, 0xadadec41, 0xa5a5f451, 0x9797a235, 0x9494a733, 0x9a9ab52f, 0x9999b029, 0x9191a839, 0x9f9fba25, 0x9b9bb62d, 0x9090ab3b, 0x9c9cbf23, 0x9393ae3d, 0x9292ad3f, 0x9696a137, 0x9898b32b, 0x9e9eb927, 0x9d9dbc21, 0x9595a431, 0x1717392e, 0x14143c28, 0x1a1a2e34, 0x19192b32, 0x11113322, 0x1f1f213e, 0x1b1b2d36, 0x10103020, 0x1c1c2438, 0x13133526, 0x12123624, 0x16163a2c, 0x18182830, 0x1e1e223c, 0x1d1d273a, 0x15153f2a, 0xf7f702f5, 0xf4f407f3, 0xfafa15ef, 0xf9f910e9, 0xf1f108f9, 0xffff1ae5, 0xfbfb16ed, 0xf0f00bfb, 0xfcfc1fe3, 0xf3f30efd, 0xf2f20dff, 0xf6f601f7, 0xf8f813eb, 0xfefe19e7, 0xfdfd1ce1, 0xf5f504f1, 0xb7b7c275, 0xb4b4c773, 0xbabad56f, 0xb9b9d069, 0xb1b1c879, 0xbfbfda65, 0xbbbbd66d, 0xb0b0cb7b, 0xbcbcdf63, 0xb3b3ce7d, 0xb2b2cd7f, 0xb6b6c177, 0xb8b8d36b, 0xbebed967, 0xbdbddc61, 0xb5b5c471, 0x0707090e, 0x04040c08, 0x0a0a1e14, 0x09091b12, 0x01010302, 0x0f0f111e, 0x0b0b1d16, 0x00000000, 0x0c0c1418, 0x03030506, 0x02020604, 0x06060a0c, 0x08081810, 0x0e0e121c, 0x0d0d171a, 0x05050f0a, 0xc7c75295, 0xc4c45793, 0xcaca458f, 0xc9c94089, 0xc1c15899, 0xcfcf4a85, 0xcbcb468d, 0xc0c05b9b, 0xcccc4f83, 0xc3c35e9d, 0xc2c25d9f, 0xc6c65197, 0xc8c8438b, 0xcece4987, 0xcdcd4c81, 0xc5c55491, 0x3737596e, 0x34345c68, 0x3a3a4e74, 0x39394b72, 0x31315362, 0x3f3f417e, 0x3b3b4d76, 0x30305060, 0x3c3c4478, 0x33335566, 0x32325664, 0x36365a6c, 0x38384870, 0x3e3e427c, 0x3d3d477a, 0x35355f6a, 0x2727694e, 0x24246c48, 0x2a2a7e54, 0x29297b52, 0x21216342, 0x2f2f715e, 0x2b2b7d56, 0x20206040, 0x2c2c7458, 0x23236546, 0x22226644, 0x26266a4c, 0x28287850, 0x2e2e725c, 0x2d2d775a, 0x25256f4a, 0x6767a9ce, 0x6464acc8, 0x6a6abed4, 0x6969bbd2, 0x6161a3c2, 0x6f6fb1de, 0x6b6bbdd6, 0x6060a0c0, 0x6c6cb4d8, 0x6363a5c6, 0x6262a6c4, 0x6666aacc, 0x6868b8d0, 0x6e6eb2dc, 0x6d6db7da, 0x6565afca, 0x87879215, 0x84849713, 0x8a8a850f, 0x89898009, 0x81819819, 0x8f8f8a05, 0x8b8b860d, 0x80809b1b, 0x8c8c8f03, 0x83839e1d, 0x82829d1f, 0x86869117, 0x8888830b, 0x8e8e8907, 0x8d8d8c01, 0x85859411, 0xe7e732d5, 0xe4e437d3, 0xeaea25cf, 0xe9e920c9, 0xe1e138d9, 0xefef2ac5, 0xebeb26cd, 0xe0e03bdb, 0xecec2fc3, 0xe3e33edd, 0xe2e23ddf, 0xe6e631d7, 0xe8e823cb, 0xeeee29c7, 0xeded2cc1, 0xe5e534d1, 0xd7d762b5, 0xd4d467b3, 0xdada75af, 0xd9d970a9, 0xd1d168b9, 0xdfdf7aa5, 0xdbdb76ad, 0xd0d06bbb, 0xdcdc7fa3, 0xd3d36ebd, 0xd2d26dbf, 0xd6d661b7, 0xd8d873ab, 0xdede79a7, 0xdddd7ca1, 0xd5d564b1, 0x5757f9ae, 0x5454fca8, 0x5a5aeeb4, 0x5959ebb2, 0x5151f3a2, 0x5f5fe1be, 0x5b5bedb6, 0x5050f0a0, 0x5c5ce4b8, 0x5353f5a6, 0x5252f6a4, 0x5656faac, 0x5858e8b0, 0x5e5ee2bc, 0x5d5de7ba, 0x5555ffaa};
bit8 S[16] = { 0x7, 0x4, 0xA, 0x9, 0x1, 0xF, 0xB, 0x0, 0xC, 0x3, 0x2, 0x6, 0x8, 0xE, 0xD, 0x5 };
bit8 S8[256] = { 0x77, 0x74, 0x7a, 0x79, 0x71, 0x7f, 0x7b, 0x70, 0x7c, 0x73, 0x72, 0x76, 0x78, 0x7e, 0x7d, 0x75, 0x47, 0x44, 0x4a, 0x49, 0x41, 0x4f, 0x4b, 0x40, 0x4c, 0x43, 0x42, 0x46, 0x48, 0x4e, 0x4d, 0x45, 0xa7, 0xa4, 0xaa, 0xa9, 0xa1, 0xaf, 0xab, 0xa0, 0xac, 0xa3, 0xa2, 0xa6, 0xa8, 0xae, 0xad, 0xa5, 0x97, 0x94, 0x9a, 0x99, 0x91, 0x9f, 0x9b, 0x90, 0x9c, 0x93, 0x92, 0x96, 0x98, 0x9e, 0x9d, 0x95, 0x17, 0x14, 0x1a, 0x19, 0x11, 0x1f, 0x1b, 0x10, 0x1c, 0x13, 0x12, 0x16, 0x18, 0x1e, 0x1d, 0x15, 0xf7, 0xf4, 0xfa, 0xf9, 0xf1, 0xff, 0xfb, 0xf0, 0xfc, 0xf3, 0xf2, 0xf6, 0xf8, 0xfe, 0xfd, 0xf5, 0xb7, 0xb4, 0xba, 0xb9, 0xb1, 0xbf, 0xbb, 0xb0, 0xbc, 0xb3, 0xb2, 0xb6, 0xb8, 0xbe, 0xbd, 0xb5, 0x7, 0x4, 0xa, 0x9, 0x1, 0xf, 0xb, 0x0, 0xc, 0x3, 0x2, 0x6, 0x8, 0xe, 0xd, 0x5, 0xc7, 0xc4, 0xca, 0xc9, 0xc1, 0xcf, 0xcb, 0xc0, 0xcc, 0xc3, 0xc2, 0xc6, 0xc8, 0xce, 0xcd, 0xc5, 0x37, 0x34, 0x3a, 0x39, 0x31, 0x3f, 0x3b, 0x30, 0x3c, 0x33, 0x32, 0x36, 0x38, 0x3e, 0x3d, 0x35, 0x27, 0x24, 0x2a, 0x29, 0x21, 0x2f, 0x2b, 0x20, 0x2c, 0x23, 0x22, 0x26, 0x28, 0x2e, 0x2d, 0x25, 0x67, 0x64, 0x6a, 0x69, 0x61, 0x6f, 0x6b, 0x60, 0x6c, 0x63, 0x62, 0x66, 0x68, 0x6e, 0x6d, 0x65, 0x87, 0x84, 0x8a, 0x89, 0x81, 0x8f, 0x8b, 0x80, 0x8c, 0x83, 0x82, 0x86, 0x88, 0x8e, 0x8d, 0x85, 0xe7, 0xe4, 0xea, 0xe9, 0xe1, 0xef, 0xeb, 0xe0, 0xec, 0xe3, 0xe2, 0xe6, 0xe8, 0xee, 0xed, 0xe5, 0xd7, 0xd4, 0xda, 0xd9, 0xd1, 0xdf, 0xdb, 0xd0, 0xdc, 0xd3, 0xd2, 0xd6, 0xd8, 0xde, 0xdd, 0xd5, 0x57, 0x54, 0x5a, 0x59, 0x51, 0x5f, 0x5b, 0x50, 0x5c, 0x53, 0x52, 0x56, 0x58, 0x5e, 0x5d, 0x55};
bit32 S8b[256] = { 0x7700, 0x7400, 0x7a00, 0x7900, 0x7100, 0x7f00, 0x7b00, 0x7000, 0x7c00, 0x7300, 0x7200, 0x7600, 0x7800, 0x7e00, 0x7d00, 0x7500, 0x4700, 0x4400, 0x4a00, 0x4900, 0x4100, 0x4f00, 0x4b00, 0x4000, 0x4c00, 0x4300, 0x4200, 0x4600, 0x4800, 0x4e00, 0x4d00, 0x4500, 0xa700, 0xa400, 0xaa00, 0xa900, 0xa100, 0xaf00, 0xab00, 0xa000, 0xac00, 0xa300, 0xa200, 0xa600, 0xa800, 0xae00, 0xad00, 0xa500, 0x9700, 0x9400, 0x9a00, 0x9900, 0x9100, 0x9f00, 0x9b00, 0x9000, 0x9c00, 0x9300, 0x9200, 0x9600, 0x9800, 0x9e00, 0x9d00, 0x9500, 0x1700, 0x1400, 0x1a00, 0x1900, 0x1100, 0x1f00, 0x1b00, 0x1000, 0x1c00, 0x1300, 0x1200, 0x1600, 0x1800, 0x1e00, 0x1d00, 0x1500, 0xf700, 0xf400, 0xfa00, 0xf900, 0xf100, 0xff00, 0xfb00, 0xf000, 0xfc00, 0xf300, 0xf200, 0xf600, 0xf800, 0xfe00, 0xfd00, 0xf500, 0xb700, 0xb400, 0xba00, 0xb900, 0xb100, 0xbf00, 0xbb00, 0xb000, 0xbc00, 0xb300, 0xb200, 0xb600, 0xb800, 0xbe00, 0xbd00, 0xb500, 0x700, 0x400, 0xa00, 0x900, 0x100, 0xf00, 0xb00, 0x0, 0xc00, 0x300, 0x200, 0x600, 0x800, 0xe00, 0xd00, 0x500, 0xc700, 0xc400, 0xca00, 0xc900, 0xc100, 0xcf00, 0xcb00, 0xc000, 0xcc00, 0xc300, 0xc200, 0xc600, 0xc800, 0xce00, 0xcd00, 0xc500, 0x3700, 0x3400, 0x3a00, 0x3900, 0x3100, 0x3f00, 0x3b00, 0x3000, 0x3c00, 0x3300, 0x3200, 0x3600, 0x3800, 0x3e00, 0x3d00, 0x3500, 0x2700, 0x2400, 0x2a00, 0x2900, 0x2100, 0x2f00, 0x2b00, 0x2000, 0x2c00, 0x2300, 0x2200, 0x2600, 0x2800, 0x2e00, 0x2d00, 0x2500, 0x6700, 0x6400, 0x6a00, 0x6900, 0x6100, 0x6f00, 0x6b00, 0x6000, 0x6c00, 0x6300, 0x6200, 0x6600, 0x6800, 0x6e00, 0x6d00, 0x6500, 0x8700, 0x8400, 0x8a00, 0x8900, 0x8100, 0x8f00, 0x8b00, 0x8000, 0x8c00, 0x8300, 0x8200, 0x8600, 0x8800, 0x8e00, 0x8d00, 0x8500, 0xe700, 0xe400, 0xea00, 0xe900, 0xe100, 0xef00, 0xeb00, 0xe000, 0xec00, 0xe300, 0xe200, 0xe600, 0xe800, 0xee00, 0xed00, 0xe500, 0xd700, 0xd400, 0xda00, 0xd900, 0xd100, 0xdf00, 0xdb00, 0xd000, 0xdc00, 0xd300, 0xd200, 0xd600, 0xd800, 0xde00, 0xdd00, 0xd500, 0x5700, 0x5400, 0x5a00, 0x5900, 0x5100, 0x5f00, 0x5b00, 0x5000, 0x5c00, 0x5300, 0x5200, 0x5600, 0x5800, 0x5e00, 0x5d00, 0x5500 };
bit32 S8c[256] = { 0x770000, 0x740000, 0x7a0000, 0x790000, 0x710000, 0x7f0000, 0x7b0000, 0x700000, 0x7c0000, 0x730000, 0x720000, 0x760000, 0x780000, 0x7e0000, 0x7d0000, 0x750000, 0x470000, 0x440000, 0x4a0000, 0x490000, 0x410000, 0x4f0000, 0x4b0000, 0x400000, 0x4c0000, 0x430000, 0x420000, 0x460000, 0x480000, 0x4e0000, 0x4d0000, 0x450000, 0xa70000, 0xa40000, 0xaa0000, 0xa90000, 0xa10000, 0xaf0000, 0xab0000, 0xa00000, 0xac0000, 0xa30000, 0xa20000, 0xa60000, 0xa80000, 0xae0000, 0xad0000, 0xa50000, 0x970000, 0x940000, 0x9a0000, 0x990000, 0x910000, 0x9f0000, 0x9b0000, 0x900000, 0x9c0000, 0x930000, 0x920000, 0x960000, 0x980000, 0x9e0000, 0x9d0000, 0x950000, 0x170000, 0x140000, 0x1a0000, 0x190000, 0x110000, 0x1f0000, 0x1b0000, 0x100000, 0x1c0000, 0x130000, 0x120000, 0x160000, 0x180000, 0x1e0000, 0x1d0000, 0x150000, 0xf70000, 0xf40000, 0xfa0000, 0xf90000, 0xf10000, 0xff0000, 0xfb0000, 0xf00000, 0xfc0000, 0xf30000, 0xf20000, 0xf60000, 0xf80000, 0xfe0000, 0xfd0000, 0xf50000, 0xb70000, 0xb40000, 0xba0000, 0xb90000, 0xb10000, 0xbf0000, 0xbb0000, 0xb00000, 0xbc0000, 0xb30000, 0xb20000, 0xb60000, 0xb80000, 0xbe0000, 0xbd0000, 0xb50000, 0x70000, 0x40000, 0xa0000, 0x90000, 0x10000, 0xf0000, 0xb0000, 0x0, 0xc0000, 0x30000, 0x20000, 0x60000, 0x80000, 0xe0000, 0xd0000, 0x50000, 0xc70000, 0xc40000, 0xca0000, 0xc90000, 0xc10000, 0xcf0000, 0xcb0000, 0xc00000, 0xcc0000, 0xc30000, 0xc20000, 0xc60000, 0xc80000, 0xce0000, 0xcd0000, 0xc50000, 0x370000, 0x340000, 0x3a0000, 0x390000, 0x310000, 0x3f0000, 0x3b0000, 0x300000, 0x3c0000, 0x330000, 0x320000, 0x360000, 0x380000, 0x3e0000, 0x3d0000, 0x350000, 0x270000, 0x240000, 0x2a0000, 0x290000, 0x210000, 0x2f0000, 0x2b0000, 0x200000, 0x2c0000, 0x230000, 0x220000, 0x260000, 0x280000, 0x2e0000, 0x2d0000, 0x250000, 0x670000, 0x640000, 0x6a0000, 0x690000, 0x610000, 0x6f0000, 0x6b0000, 0x600000, 0x6c0000, 0x630000, 0x620000, 0x660000, 0x680000, 0x6e0000, 0x6d0000, 0x650000, 0x870000, 0x840000, 0x8a0000, 0x890000, 0x810000, 0x8f0000, 0x8b0000, 0x800000, 0x8c0000, 0x830000, 0x820000, 0x860000, 0x880000, 0x8e0000, 0x8d0000, 0x850000, 0xe70000, 0xe40000, 0xea0000, 0xe90000, 0xe10000, 0xef0000, 0xeb0000, 0xe00000, 0xec0000, 0xe30000, 0xe20000, 0xe60000, 0xe80000, 0xee0000, 0xed0000, 0xe50000, 0xd70000, 0xd40000, 0xda0000, 0xd90000, 0xd10000, 0xdf0000, 0xdb0000, 0xd00000, 0xdc0000, 0xd30000, 0xd20000, 0xd60000, 0xd80000, 0xde0000, 0xdd0000, 0xd50000, 0x570000, 0x540000, 0x5a0000, 0x590000, 0x510000, 0x5f0000, 0x5b0000, 0x500000, 0x5c0000, 0x530000, 0x520000, 0x560000, 0x580000, 0x5e0000, 0x5d0000, 0x550000 };

__device__ bit32 arithmeticRightShift(bit32 x, bit32 n) { return (x >> n) | (x << (-n & 31)); }

__device__ bit32 arithmeticRightShiftBytePerm(bit32 x, bit32 n) { return __byte_perm(x, x, n); }
void gmix_column(unsigned char* r) {
    unsigned char a[4];
    unsigned char b[4];
    unsigned char c;
    unsigned char h;
    /* The array 'a' is simply a copy of the input array 'r'
     * The array 'b' is each element of the array 'a' multiplied by 2
     * in Rijndael's Galois field
     * a[n] ^ b[n] is element n multiplied by 3 in Rijndael's Galois field */
    for (c = 0; c < 4; c++) {
        a[c] = r[c];
        /* h is set to 0x01 if the high bit of r[c] is set, 0x00 otherwise */
        h = r[c] >> 7;    /* logical right shift, thus shifting in zeros */
        b[c] = r[c] << 1; /* implicitly removes high bit because b[c] is an 8-bit char, so we xor by 0x1b and not 0x11b in the next line */
        b[c] ^= h * 0x1B; /* Rijndael's Galois field */
    }
    r[0] = b[0] ^ a[3] ^ a[2] ^ b[1] ^ a[1]; /* 2 * a0 + a3 + a2 + 3 * a1 */
    r[1] = b[1] ^ a[0] ^ a[3] ^ b[2] ^ a[2]; /* 2 * a1 + a0 + a3 + 3 * a2 */
    r[2] = b[2] ^ a[1] ^ a[0] ^ b[3] ^ a[3]; /* 2 * a2 + a1 + a0 + 3 * a3 */
    r[3] = b[3] ^ a[2] ^ a[1] ^ b[0] ^ a[0]; /* 2 * a3 + a2 + a1 + 3 * a0 */
}
void calculate_tables() {
    bit8 r[4] = { 0 };
    bit32 result;
    for (bit32 i = 0; i < 256; i++) {
        r[0] = S[i & 0xf] ^ (S[i >> 4] << 4);
        r[1] = 0; r[2] = 0; r[3] = 0;
        gmix_column(r);
        result = (r[0]<<24) ^ (r[1] << 16) ^ (r[2] << 8) ^ (r[3] << 0);
        printf("0x%08x, ", result);
    }
    printf("\n");
    for (bit32 i = 0; i < 256; i++) {
        r[1] = S[i & 0xf] ^ (S[i >> 4] << 4);
        r[0] = 0; r[2] = 0; r[3] = 0;
        gmix_column(r);
        result = (r[0] << 24) ^ (r[1] << 16) ^ (r[2] << 8) ^ (r[3] << 0);
        printf("0x%08x, ", result);
    }
    printf("\n");
    for (bit32 i = 0; i < 256; i++) {
        r[2] = S[i & 0xf] ^ (S[i >> 4] << 4);
        r[0] = 0; r[1] = 0; r[3] = 0;
        gmix_column(r);
        result = (r[0] << 24) ^ (r[1] << 16) ^ (r[2] << 8) ^ (r[3] << 0);
        printf("0x%08x, ", result);
    }
    printf("\n");
    for (bit32 i = 0; i < 256; i++) {
        r[3] = S[i & 0xf] ^ (S[i >> 4] << 4);
        r[0] = 0; r[1] = 0; r[2] = 0;
        gmix_column(r);
        result = (r[0] << 24) ^ (r[1] << 16) ^ (r[2] << 8) ^ (r[3] << 0);
        printf("0x%08x, ", result);
    }
    printf("\n");
}
void generate_s8() {
    bit8 temp;
    for (int i = 0; i < 256; i++) {
        temp = S[i & 0xf] ^ (S[i >> 4] << 4);
        printf("0x%x, ",temp);
    }
}
void KLEIN64() {
	bit8 plaintext[8] = { 0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF }, input[8] = { 0x0 }, output[8] = { 0x0 };
    bit8 key[8] = { 0x00 }, temp[8] = { 0x0 };
	for (int i = 0; i < 8; i++) input [i] = plaintext[i];
     for (int j = 1; j <= 12; j++) {
        for (int i = 0; i < 8; i++) input[i] ^= key[i];
        for (int i = 0; i < 8; i++) { output[i] = S[input[i] & 0xf] ^ (S[input[i] >> 4] << 4); }
        for (int i = 0; i < 8; i++) input[(i+2)%8] = output[i];
        for (int i = 7; i >= 0; i--) printf("%02x", key[i]);    printf("\n");
        output[0] = input[3];
        output[1] = input[2];
        output[2] = input[1];
        output[3] = input[0];

        output[4] = input[7];
        output[5] = input[6];
        output[6] = input[5];
        output[7] = input[4];
        gmix_column(output);     gmix_column(output + 4);
        input[0] = output[3];
        input[1] = output[2];
        input[2] = output[1];
        input[3] = output[0];

        input[4] = output[7];
        input[5] = output[6];
        input[6] = output[5];
        input[7] = output[4];
 
        temp[0] = key[3];
        temp[1] = key[0];
        temp[2] = key[1];
        temp[3] = key[2];
        temp[4] = key[7];
        temp[5] = key[4];
        temp[6] = key[5];
        temp[7] = key[6];

        temp[4] ^= temp[0];
        temp[5] ^= temp[1];
        temp[6] ^= temp[2];
        temp[7] ^= temp[3];

        key[0] = temp[4];
        key[1] = temp[5];
        key[2] = temp[6];
        key[3] = temp[7];
        key[4] = temp[0];
        key[5] = temp[1];
        key[6] = temp[2];
        key[7] = temp[3];
        key[5] ^= j;
        key[1] = S[key[1] & 0xf] ^ (S[key[1] >> 4] << 4);
        key[2] = S[key[2] & 0xf] ^ (S[key[2] >> 4] << 4);
    }
    for (int i = 0; i < 8; i++) input[i] ^= key[i];
    for (int i = 7; i >=0; i--) printf("%02x",input[i]);    printf("\n");
}
void KLEIN64_table_based() {
    bit32 plaintext0 = 0xFFFFFFFF; // I image as the 64 bit is located as "plaintext1 plaintext0"
    bit32 plaintext1 = 0xFFFFFFFF;
    bit32 key0 = 0x0;
    bit32 key1 = 0x0;
    bit32 temp1 = 0x0, temp0 = 0x0;

    for (bit32 j = 1; j <= 12; j++) {
        temp1 = plaintext1 ^ key1;
        temp0 = plaintext0 ^ key0;

        plaintext0 = T3[(temp1 & 0x00FF0000) >> 16] ^ T2[(temp1 & 0xFF000000) >> 24] ^ T1[temp0 & 0x000000FF] ^ T0[(temp0 & 0x0000FF00) >> 8];
        plaintext1 = T3[(temp0 & 0x00FF0000) >> 16] ^ T2[(temp0 & 0xFF000000) >> 24] ^ T1[temp1 & 0x000000FF] ^ T0[(temp1 & 0x0000FF00) >> 8];
        printf("%08x%08x\n", key1, key0);

        key0 = (key0 << 8) ^ (key0 >> 24);
        key1 = (key1 << 8) ^ (key1 >> 24);
        key1 ^= key0;

        temp1 = key0;
        key0 = key1;
        key1 = temp1;

        key1 ^= (j << 8);
        key0 = (key0 & 0xFF0000FF) ^ (S[(key0 & 0x00000F00)>>8]<<8) ^ (S[(key0 & 0x0000F000) >> 12] << 12) ^ (S[(key0 & 0x000F0000) >> 16] << 16) ^ (S[(key0 & 0x00F00000) >> 20] << 20);
    }
    plaintext1 = plaintext1 ^ key1;
    plaintext0 = plaintext0 ^ key0;
    printf("%08x %08x\n",plaintext1, plaintext0);
}
__global__ void KLEIN64ExhaustiveSearch(bit32 pt1, bit32 pt0, bit32 ct1, bit32 ct0, bit32 *T0G, bit32 *T1G, bit32 *T2G, bit32 *T3G, bit8* SG ) {
    bit32 threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ bit32 T0S[256];
    __shared__ bit32 T1S[256];
    __shared__ bit32 T2S[256];
    __shared__ bit32 T3S[256];
    __shared__ bit8 SS[16];
    if (threadIdx.x < 256) {
        if (threadIdx.x < 16) SS[threadIdx.x] = SG[threadIdx.x];
        T0S[threadIdx.x] = T0G[threadIdx.x];
        T1S[threadIdx.x] = T1G[threadIdx.x];
        T2S[threadIdx.x] = T2G[threadIdx.x];
        T3S[threadIdx.x] = T3G[threadIdx.x];
    }
    __syncthreads();
    bit32 temp0, temp1, plaintext1, plaintext0;
    bit32 key1 = threadIndex, key0 = 0;
    bit32 ciphertext1 = ct1;
    bit32 ciphertext0 = ct0;
    for (int i = 0; i < 1024*32; i++) {
        plaintext1 = pt1;
        plaintext0 = pt0;
        key1 = threadIndex;
        key0 = i;
        for (bit32 j = 1; j <= 12; j++) {
            temp1 = plaintext1 ^ key1;
            temp0 = plaintext0 ^ key0;

            plaintext0 = T3S[(temp1 & 0x00FF0000) >> 16] ^ T2S[(temp1 & 0xFF000000) >> 24] ^ T1S[temp0 & 0x000000FF] ^ T0S[(temp0 & 0x0000FF00) >> 8];
            plaintext1 = T3S[(temp0 & 0x00FF0000) >> 16] ^ T2S[(temp0 & 0xFF000000) >> 24] ^ T1S[temp1 & 0x000000FF] ^ T0S[(temp1 & 0x0000FF00) >> 8];

            key0 = (key0 << 8) ^ (key0 >> 24);
            key1 = (key1 << 8) ^ (key1 >> 24);
            key1 ^= key0;

            temp1 = key0;
            key0 = key1;
            key1 = temp1;

            key1 ^= (j << 8);
            key0 = (key0 & 0xFF0000FF) ^ (SS[(key0 & 0x00000F00) >> 8] << 8) ^ (SS[(key0 & 0x0000F000) >> 12] << 12) ^ (SS[(key0 & 0x000F0000) >> 16] << 16) ^ (SS[(key0 & 0x00F00000) >> 20] << 20);
        }
        plaintext1 = plaintext1 ^ key1;
        plaintext0 = plaintext0 ^ key0;
        if (plaintext1 == ciphertext1)
            if (plaintext0 == ciphertext0)
                printf("The secret key is %08x%08x\n", threadIndex, i);
    }

}
__global__ void KLEIN64ExhaustiveSearch32Copies(bit32 pt1, bit32 pt0, bit32 ct1, bit32 ct0, bit32* T0G, bit32* T1G, bit32* T2G, bit32* T3G, bit8* SG) {
    bit32 threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int warpThreadIndex = threadIdx.x & 31;
    __shared__ bit32 T0S[256][32];
    __shared__ bit32 T1S[256];
    __shared__ bit32 T2S[256];
    __shared__ bit32 T3S[256];
    __shared__ bit8 SS[16];
    if (threadIdx.x < 256) {
        if (threadIdx.x < 16) SS[threadIdx.x] = SG[threadIdx.x];
        for (int i=0;i<32;i++) T0S[threadIdx.x][i] = T0G[threadIdx.x];
        T1S[threadIdx.x] = T1G[threadIdx.x];
        T2S[threadIdx.x] = T2G[threadIdx.x];
        T3S[threadIdx.x] = T3G[threadIdx.x];
    }
    __syncthreads();
    bit32 temp0, temp1,j;
    bit32 ciphertext1 = ct1;
    bit32 ciphertext0 = ct0;
    bit32 plaintext1, plaintext0, key1, key0;
    for (int i = 0; i < 1024*32; i++) {
        plaintext1 = pt1;
        plaintext0 = pt0;
        key1 = threadIndex;
        key0 = i;
        for (j = 1; j <= 12; j++) {
            temp1 = plaintext1 ^ key1;
            temp0 = plaintext0 ^ key0;

            plaintext0 = T3S[(temp1 & 0x00FF0000) >> 16] ^ T2S[(temp1 & 0xFF000000) >> 24] ^ T1S[temp0 & 0x000000FF] ^ T0S[(temp0 & 0x0000FF00) >> 8][warpThreadIndex];
            plaintext1 = T3S[(temp0 & 0x00FF0000) >> 16] ^ T2S[(temp0 & 0xFF000000) >> 24] ^ T1S[temp1 & 0x000000FF] ^ T0S[(temp1 & 0x0000FF00) >> 8][warpThreadIndex];

            key0 = (key0 << 8) ^ (key0 >> 24);
            key1 = (key1 << 8) ^ (key1 >> 24);
            key1 ^= key0;

            temp1 = key0;
            key0 = key1;
            key1 = temp1;

            key1 ^= (j << 8);
            key0 = (key0 & 0xFF0000FF) ^ (SS[(key0 & 0x00000F00) >> 8] << 8) ^ (SS[(key0 & 0x0000F000) >> 12] << 12) ^ (SS[(key0 & 0x000F0000) >> 16] << 16) ^ (SS[(key0 & 0x00F00000) >> 20] << 20);
        }
        plaintext1 = plaintext1 ^ key1;
        plaintext0 = plaintext0 ^ key0;
        if (plaintext1 == ciphertext1)
            if (plaintext0 == ciphertext0)
                printf("The secret key is %08x%08x\n", threadIndex, i);
    }
}
__global__ void KLEIN64ExhaustiveSearch32CopiesS8(bit32 pt1, bit32 pt0, bit32 ct1, bit32 ct0, bit32* T0G, bit32* T1G, bit32* T2G, bit32* T3G, bit8* SG) {
    bit32 threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int warpThreadIndex = threadIdx.x & 31;
    __shared__ bit32 T0S[256][32];
    __shared__ bit32 T1S[256];
    __shared__ bit32 T2S[256];
    __shared__ bit32 T3S[256];
    __shared__ bit8 SS[256];
    if (threadIdx.x < 256) {
        SS[threadIdx.x] = SG[threadIdx.x];
        for (int i = 0; i < 32; i++) T0S[threadIdx.x][i] = T0G[threadIdx.x];
        T1S[threadIdx.x] = T1G[threadIdx.x];
        T2S[threadIdx.x] = T2G[threadIdx.x];
        T3S[threadIdx.x] = T3G[threadIdx.x];
    }
    __syncthreads();
    bit32 temp0, temp1, j;
    bit32 ciphertext1 = ct1;
    bit32 ciphertext0 = ct0;
    bit32 plaintext1, plaintext0, key1, key0;
    for (int i = 0; i < 1024 * 32; i++) {
        plaintext1 = pt1;
        plaintext0 = pt0;
        key1 = threadIndex;
        key0 = i;
        for (j = 1; j <= 12; j++) {
            temp1 = plaintext1 ^ key1;
            temp0 = plaintext0 ^ key0;

            plaintext0 = T3S[(temp1 & 0x00FF0000) >> 16] ^ T2S[(temp1 & 0xFF000000) >> 24] ^ T1S[temp0 & 0x000000FF] ^ T0S[(temp0 & 0x0000FF00) >> 8][warpThreadIndex];
            plaintext1 = T3S[(temp0 & 0x00FF0000) >> 16] ^ T2S[(temp0 & 0xFF000000) >> 24] ^ T1S[temp1 & 0x000000FF] ^ T0S[(temp1 & 0x0000FF00) >> 8][warpThreadIndex];

            key0 = (key0 << 8) ^ (key0 >> 24);
            key1 = (key1 << 8) ^ (key1 >> 24);
            key1 ^= key0;

            temp1 = key0;
            key0 = key1;
            key1 = temp1;

            key1 ^= (j << 8);
            key0 = (key0 & 0xFF0000FF) ^ (SS[(key0 & 0x0000FF00) >> 8] << 8) ^ (SS[(key0 & 0x00FF0000) >> 16] << 16) ;
        }
        plaintext1 = plaintext1 ^ key1;
        plaintext0 = plaintext0 ^ key0;
        if (plaintext1 == ciphertext1)
            if (plaintext0 == ciphertext0)
                printf("The secret key is %08x%08x\n", threadIndex, i);
    }
}
__global__ void KLEIN64ExhaustiveSearch32CopiesS8SingleTable(bit32 pt1, bit32 pt0, bit32 ct1, bit32 ct0, bit32* T0G, bit8* SG) {
    bit32 threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int warpThreadIndex = threadIdx.x & 31;
    __shared__ bit32 T0S[256][32];
    __shared__ bit8 SS[256];
    if (threadIdx.x < 256) {
        SS[threadIdx.x] = SG[threadIdx.x];
        for (int i = 0; i < 32; i++) T0S[threadIdx.x][i] = T0G[threadIdx.x];
    }
    __syncthreads();
    bit32 temp0, temp1, j;
    bit32 ciphertext1 = ct1;
    bit32 ciphertext0 = ct0;
    bit32 plaintext1, plaintext0, key1, key0;

    for (int i = 0; i < 1024 * 32; i++) {
        plaintext1 = pt1;
        plaintext0 = pt0;
        key1 = threadIndex;
        key0 = i;
#pragma unroll
        for (j = 1; j <= 12; j++) {
            temp1 = plaintext1 ^ key1;
            temp0 = plaintext0 ^ key0;

            plaintext0 = arithmeticRightShift(T0S[(temp1 & 0x00FF0000) >> 16][warpThreadIndex],24) ^ arithmeticRightShift(T0S[(temp1 & 0xFF000000) >> 24][warpThreadIndex],16) ^ arithmeticRightShift(T0S[temp0 & 0x000000FF][warpThreadIndex],8) ^ T0S[(temp0 & 0x0000FF00) >> 8][warpThreadIndex];
            plaintext1 = arithmeticRightShift(T0S[(temp0 & 0x00FF0000) >> 16][warpThreadIndex],24) ^ arithmeticRightShift(T0S[(temp0 & 0xFF000000) >> 24][warpThreadIndex],16) ^ arithmeticRightShift(T0S[temp1 & 0x000000FF][warpThreadIndex],8) ^ T0S[(temp1 & 0x0000FF00) >> 8][warpThreadIndex];

            key0 = arithmeticRightShift(key0, 24);
            key1 = arithmeticRightShift(key1, 24);
            key1 ^= key0;

            temp1 = key0;
            key0 = key1;
            key1 = temp1;

            key1 ^= (j << 8);
            key0 = (key0 & 0xFF0000FF) ^ (SS[(key0 & 0x0000FF00) >> 8] << 8) ^ (SS[(key0 & 0x00FF0000) >> 16] << 16);
        }
        plaintext1 = plaintext1 ^ key1;
        plaintext0 = plaintext0 ^ key0;
        if (plaintext1 == ciphertext1)
            if (plaintext0 == ciphertext0)
                printf("The secret key is %08x%08x\n", threadIndex, i);
    }
}
__global__ void KLEIN64ExhaustiveSearch32CopiesS8SingleTableShift(bit32 pt1, bit32 pt0, bit32 ct1, bit32 ct0, bit32* T0G, bit32* SG1, bit32* SG2) {
    bit32 threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int warpThreadIndex = threadIdx.x & 31;
    __shared__ bit32 T0S[256][32];
    __shared__ bit32 S1[256];
    __shared__ bit32 S2[256];
    if (threadIdx.x < 256) {
        S1[threadIdx.x] = SG1[threadIdx.x];
        S2[threadIdx.x] = SG2[threadIdx.x];
        for (int i = 0; i < 32; i++) T0S[threadIdx.x][i] = T0G[threadIdx.x];
    }
    __syncthreads();
    bit32 temp0, temp1, j;
    bit32 ciphertext1 = ct1;
    bit32 ciphertext0 = ct0;
    bit32 plaintext1, plaintext0, key1, key0;

    for (int i = 0; i < 1024 * 32; i++) {
        plaintext1 = pt1;
        plaintext0 = pt0;
        key1 = threadIndex;
        key0 = i;
#pragma unroll
        for (j = 1; j <= 12; j++) {
            temp1 = plaintext1 ^ key1;
            temp0 = plaintext0 ^ key0;

            plaintext0 = arithmeticRightShift(T0S[(temp1 & 0x00FF0000) >> 16][warpThreadIndex], 24) ^ arithmeticRightShift(T0S[(temp1 & 0xFF000000) >> 24][warpThreadIndex], 16) ^ arithmeticRightShift(T0S[temp0 & 0x000000FF][warpThreadIndex], 8) ^ T0S[(temp0 & 0x0000FF00) >> 8][warpThreadIndex];
            plaintext1 = arithmeticRightShift(T0S[(temp0 & 0x00FF0000) >> 16][warpThreadIndex], 24) ^ arithmeticRightShift(T0S[(temp0 & 0xFF000000) >> 24][warpThreadIndex], 16) ^ arithmeticRightShift(T0S[temp1 & 0x000000FF][warpThreadIndex], 8) ^ T0S[(temp1 & 0x0000FF00) >> 8][warpThreadIndex];

            key0 = arithmeticRightShift(key0, 24);
            key1 = arithmeticRightShift(key1, 24);
            key1 ^= key0;

            temp1 = key0;
            key0 = key1;
            key1 = temp1;

            key1 ^= (j << 8);
            key0 = (key0 & 0xFF0000FF) ^ (S1[(key0 & 0x0000FF00) >> 8]) ^ (S2[(key0 & 0x00FF0000) >> 16]);
        }
        plaintext1 = plaintext1 ^ key1;
        plaintext0 = plaintext0 ^ key0;
        if (plaintext1 == ciphertext1)
            if (plaintext0 == ciphertext0)
                printf("The secret key is %08x%08x\n", threadIndex, i);
    }
}
__global__ void KLEIN64ExhaustiveSearch32CopiesS8SingleTableShiftBytePerm(bit32 pt1, bit32 pt0, bit32 ct1, bit32 ct0, bit32* T0G, bit32* SG1, bit32* SG2) {
    bit32 threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int warpThreadIndex = threadIdx.x & 31;
    __shared__ bit32 T0S[256][32];
    __shared__ bit32 S1[256];
    __shared__ bit32 S2[256];
    if (threadIdx.x < 256) {
        S1[threadIdx.x] = SG1[threadIdx.x];
        S2[threadIdx.x] = SG2[threadIdx.x];
        for (int i = 0; i < 32; i++) T0S[threadIdx.x][i] = T0G[threadIdx.x];
    }
    __syncthreads();
    bit32 temp0, temp1, j;
    bit32 ciphertext1 = ct1;
    bit32 ciphertext0 = ct0;
    bit32 plaintext1, plaintext0, key1, key0;

    for (int i = 0; i < 1024 * 32; i++) {
        plaintext1 = pt1;
        plaintext0 = pt0;
        key1 = threadIndex;
        key0 = i;
#pragma unroll
        for (j = 1; j <= 12; j++) {
            temp1 = plaintext1 ^ key1;
            temp0 = plaintext0 ^ key0;
            plaintext0 = arithmeticRightShiftBytePerm(T0S[(temp1 & 0x00FF0000) >> 16][warpThreadIndex], SHIFT_3_RIGHT) ^ arithmeticRightShiftBytePerm(T0S[(temp1 & 0xFF000000) >> 24][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(T0S[temp0 & 0x000000FF][warpThreadIndex], SHIFT_1_RIGHT) ^ T0S[(temp0 & 0x0000FF00) >> 8][warpThreadIndex];
            plaintext1 = arithmeticRightShiftBytePerm(T0S[(temp0 & 0x00FF0000) >> 16][warpThreadIndex], SHIFT_3_RIGHT) ^ arithmeticRightShiftBytePerm(T0S[(temp0 & 0xFF000000) >> 24][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(T0S[temp1 & 0x000000FF][warpThreadIndex], SHIFT_1_RIGHT) ^ T0S[(temp1 & 0x0000FF00) >> 8][warpThreadIndex];
            key0 = arithmeticRightShiftBytePerm(key0, SHIFT_3_RIGHT);
            key1 = arithmeticRightShiftBytePerm(key1, SHIFT_3_RIGHT);
            key1 ^= key0;
            temp1 = key0;            key0 = key1;            key1 = temp1;
            key1 ^= (j << 8);
            key0 = (key0 & 0xFF0000FF) ^ (S1[arithmeticRightShiftBytePerm(key0 & 0x0000FF00, SHIFT_1_RIGHT)]) ^ (S2[arithmeticRightShiftBytePerm(key0 & 0x00FF0000, SHIFT_2_RIGHT)]);
        }
        plaintext1 = plaintext1 ^ key1;
        plaintext0 = plaintext0 ^ key0;
        if (plaintext1 == ciphertext1)
            if (plaintext0 == ciphertext0)
                printf("The secret key is %08x%08x\n", threadIndex, i);
    }
}
__global__ void KLEIN96ExhaustiveSearch32CopiesS8SingleTable(bit32 pt1, bit32 pt0, bit32 ct1, bit32 ct0, bit32* T0G, bit8* SG) {
    bit64 threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int warpThreadIndex = threadIdx.x & 31;
    __shared__ bit32 T0S[256][32];
    __shared__ bit8 SS[256];
    if (threadIdx.x < 256) {
        SS[threadIdx.x] = SG[threadIdx.x];
        for (int i = 0; i < 32; i++) T0S[threadIdx.x][i] = T0G[threadIdx.x];
    }
    __syncthreads();
    bit32 temp0, temp1, j;
    bit32 ciphertext1 = ct1;
    bit32 ciphertext0 = ct0;
    bit32 plaintext1, plaintext0;
    bit64 key1, key0, temp2;

    for (bit64 i = 0; i < 1024 * 32; i++) {
        plaintext1 = pt1;
        plaintext0 = pt0;
        key1 = threadIndex;
        key0 = i;
#pragma unroll
        for (j = 1; j <= 20; j++) {
            temp1 = plaintext1 ^ (key1 >> 16);
            temp0 = plaintext0 ^ (key1 << 16) ^ (key0 >> 32);

            plaintext0 = arithmeticRightShiftBytePerm(T0S[(temp1 & 0x00FF0000) >> 16][warpThreadIndex], SHIFT_3_RIGHT) ^ arithmeticRightShiftBytePerm(T0S[(temp1 & 0xFF000000) >> 24][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(T0S[temp0 & 0x000000FF][warpThreadIndex], SHIFT_1_RIGHT) ^ T0S[(temp0 & 0x0000FF00) >> 8][warpThreadIndex];
            plaintext1 = arithmeticRightShiftBytePerm(T0S[(temp0 & 0x00FF0000) >> 16][warpThreadIndex], SHIFT_3_RIGHT) ^ arithmeticRightShiftBytePerm(T0S[(temp0 & 0xFF000000) >> 24][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(T0S[temp1 & 0x000000FF][warpThreadIndex], SHIFT_1_RIGHT) ^ T0S[(temp1 & 0x0000FF00) >> 8][warpThreadIndex];


            key0 = ((key0 << 8) ^ (key0 >> 40)) & 0x0000FFFFFFFFFFFF;
            key1 = ((key1 << 8) ^ (key1 >> 40)) & 0x0000FFFFFFFFFFFF;
            key1 ^= key0;

            temp2 = key0;
            key0 = key1;
            key1 = temp2;

            key1 ^= (j << 24);
            key0 = (key0 & 0xFF0000FFFFFF) ^ (SS[(key0 & 0x0000FF000000) >> 24] << 24) ^ (bit64(SS[(key0 & 0x00FF00000000) >> 32]) << 32);
        }
        plaintext1 = plaintext1 ^ (key1>>16);
        plaintext0 = plaintext0 ^ (key1<<16) ^ (key0>>32);
        if (plaintext1 == ciphertext1)
            if (plaintext0 == ciphertext0)
                printf("The secret key is %08llx%08llx\n", threadIndex, i);
    }
}
__global__ void KLEIN80ExhaustiveSearch32CopiesS8SingleTable(bit32 pt1, bit32 pt0, bit32 ct1, bit32 ct0, bit32* T0G, bit8* SG) {
    bit64 threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int warpThreadIndex = threadIdx.x & 31;
    __shared__ bit32 T0S[256][32];
    __shared__ bit8 SS[256];
    if (threadIdx.x < 256) {
        SS[threadIdx.x] = SG[threadIdx.x];
        for (int i = 0; i < 32; i++) T0S[threadIdx.x][i] = T0G[threadIdx.x];
    }
    __syncthreads();
    bit32 temp0, temp1, j;
    bit32 ciphertext1 = ct1;
    bit32 ciphertext0 = ct0;
    bit32 plaintext1, plaintext0;
    bit64 key1, key0, temp2;

    for (bit64 i = 0; i < 1024 * 32; i++) {
        plaintext1 = pt1;
        plaintext0 = pt0;
        key1 = threadIndex;
        key0 = i;
#pragma unroll
        for (j = 1; j <= 16; j++) {
            temp1 = plaintext1 ^ (key1 >> 8);
            temp0 = plaintext0 ^ (key1 << 24) ^ (key0 >> 16);

 //           plaintext0 = arithmeticRightShift(T0S[(temp1 & 0x00FF0000) >> 16][warpThreadIndex], 24) ^ arithmeticRightShift(T0S[(temp1 & 0xFF000000) >> 24][warpThreadIndex], 16) ^ arithmeticRightShift(T0S[temp0 & 0x000000FF][warpThreadIndex], 8) ^ T0S[(temp0 & 0x0000FF00) >> 8][warpThreadIndex];
 //           plaintext1 = arithmeticRightShift(T0S[(temp0 & 0x00FF0000) >> 16][warpThreadIndex], 24) ^ arithmeticRightShift(T0S[(temp0 & 0xFF000000) >> 24][warpThreadIndex], 16) ^ arithmeticRightShift(T0S[temp1 & 0x000000FF][warpThreadIndex], 8) ^ T0S[(temp1 & 0x0000FF00) >> 8][warpThreadIndex];
            plaintext0 = arithmeticRightShiftBytePerm(T0S[(temp1 & 0x00FF0000) >> 16][warpThreadIndex], SHIFT_3_RIGHT) ^ arithmeticRightShiftBytePerm(T0S[(temp1 & 0xFF000000) >> 24][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(T0S[temp0 & 0x000000FF][warpThreadIndex], SHIFT_1_RIGHT) ^ T0S[(temp0 & 0x0000FF00) >> 8][warpThreadIndex];
            plaintext1 = arithmeticRightShiftBytePerm(T0S[(temp0 & 0x00FF0000) >> 16][warpThreadIndex], SHIFT_3_RIGHT) ^ arithmeticRightShiftBytePerm(T0S[(temp0 & 0xFF000000) >> 24][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(T0S[temp1 & 0x000000FF][warpThreadIndex], SHIFT_1_RIGHT) ^ T0S[(temp1 & 0x0000FF00) >> 8][warpThreadIndex];

            key0 = ((key0 << 8) ^ (key0 >> 32)) & 0x000000FFFFFFFFFF;
            key1 = ((key1 << 8) ^ (key1 >> 32)) & 0x000000FFFFFFFFFF;
            key1 ^= key0;

            temp2 = key0;
            key0 = key1;
            key1 = temp2;

            key1 ^= (j << 16);
            key0 = (key0 & 0xFF0000FFFF) ^ (SS[(key0 & 0x0000FF0000) >> 16] << 16) ^ (SS[(key0 & 0x00FF000000) >> 24] << 24);
        }
        plaintext1 = plaintext1 ^ (key1 >> 8);
        plaintext0 = plaintext0 ^ (key1 << 24) ^ (key0 >> 16);
        if (plaintext1 == ciphertext1)
            if (plaintext0 == ciphertext0)
                printf("The secret key is %08llx%08llx\n", threadIndex, i);
    }
}
__global__ void KLEIN64ExhaustiveSearch32CopiesS8SingleTableS(bit32 pt1, bit32 pt0, bit32 ct1, bit32 ct0, bit32* T0G, bit8* SG) {
    bit32 threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int warpThreadIndex = threadIdx.x & 31;
    __shared__ bit32 T0S[256][32];
    __shared__ bit8 SS[64][32][4];
    if (threadIdx.x < 256) {
        for (int i = 0; i < 32; i++) {
            T0S[threadIdx.x][i] = T0G[threadIdx.x];
            SS[threadIdx.x / 4][i][threadIdx.x % 4] = SG[threadIdx.x];
        }
    }
    __syncthreads();
    bit32 temp0, temp1, j;
    bit32 ciphertext1 = ct1;
    bit32 ciphertext0 = ct0;
    bit32 plaintext1, plaintext0, key1, key0;
    for (int i = 0; i < 1024 * 32; i++) {
        plaintext1 = pt1;
        plaintext0 = pt0;
        key1 = threadIndex;
        key0 = i;
        for (j = 1; j <= 12; j++) {
            temp1 = plaintext1 ^ key1;
            temp0 = plaintext0 ^ key0;

            plaintext0 = arithmeticRightShift(T0S[(temp1 & 0x00FF0000) >> 16][warpThreadIndex], 24) ^ arithmeticRightShift(T0S[(temp1 & 0xFF000000) >> 24][warpThreadIndex], 16) ^ arithmeticRightShift(T0S[temp0 & 0x000000FF][warpThreadIndex], 8) ^ T0S[(temp0 & 0x0000FF00) >> 8][warpThreadIndex];
            plaintext1 = arithmeticRightShift(T0S[(temp0 & 0x00FF0000) >> 16][warpThreadIndex], 24) ^ arithmeticRightShift(T0S[(temp0 & 0xFF000000) >> 24][warpThreadIndex], 16) ^ arithmeticRightShift(T0S[temp1 & 0x000000FF][warpThreadIndex], 8) ^ T0S[(temp1 & 0x0000FF00) >> 8][warpThreadIndex];

            key0 = (key0 << 8) ^ (key0 >> 24);
            key1 = (key1 << 8) ^ (key1 >> 24);
            key1 ^= key0;

            temp1 = key0;
            key0 = key1;
            key1 = temp1;

            key1 ^= (j << 8);
            key0 = (key0 & 0xFF0000FF) ^ (SS[((key0 & 0x0000FF00) >> 8)/4][warpThreadIndex][((key0 & 0x0000FF00) >> 8) % 4] << 8) ^ (SS[((key0 & 0x00FF0000) >> 16)/4][warpThreadIndex][((key0 & 0x00FF0000) >> 16) % 4] << 16);
        }
        plaintext1 = plaintext1 ^ key1;
        plaintext0 = plaintext0 ^ key0;
        if (plaintext1 == ciphertext1)
            if (plaintext0 == ciphertext0)
                printf("The secret key is %08x%08x\n", threadIndex, i);
    }
}
void ExhaustiveSearch() {
    bit32 plaintext0 = 0xFFFFFFFF; // I imagine as the 64 bit is located as "plaintext1 plaintext0"
    bit32 plaintext1 = 0xFFFFFFFF;
    bit32 ciphertext0 = 0x14722bbe; 
    bit32 ciphertext1 = 0xcdc0b51f;
    bit32 ciphertext2 = 0x3D8E8E36;
    bit32 ciphertext3 = 0xDB9FA7D3;
    bit32 ciphertext4 = 0x1A53A431;
    bit32 ciphertext5 = 0x6677E20D;

    // Allocate Tables
    bit32* t0, * t1, * t2, * t3, *s8b, *s8c;    bit8* s4; bit8* s8;
    hipMallocManaged(&t0, 256 * sizeof(bit32));
    hipMallocManaged(&t1, 256 * sizeof(bit32));
    hipMallocManaged(&t2, 256 * sizeof(bit32));
    hipMallocManaged(&t3, 256 * sizeof(bit32));
    hipMallocManaged(&s8b, 256 * sizeof(bit32));
    hipMallocManaged(&s8c, 256 * sizeof(bit32));
    hipMallocManaged(&s4, 16 * sizeof(bit8));
    hipMallocManaged(&s8, 256 * sizeof(bit8));
    for (int i = 0; i < 256; i++) {
        t0[i] = T0[i];
        t1[i] = T1[i];
        t2[i] = T2[i];
        t3[i] = T3[i];
        s8[i] = S8[i];
        s8b[i] = S8b[i];
        s8c[i] = S8c[i];
    }
    for (int i = 0; i < 16; i++) s4[i] = S[i];

    clock_t beginTime = clock();
//    KLEIN64ExhaustiveSearch << <BLOCKS, THREADS >> > (plaintext1, plaintext0, ciphertext1, ciphertext0,t0,t1,t2,t3,s4);
//    KLEIN64ExhaustiveSearch32Copies << <BLOCKS, THREADS >> > (plaintext1, plaintext0, ciphertext1, ciphertext0, t0, t1, t2, t3, s4);
//    KLEIN64ExhaustiveSearch32CopiesS8 << <BLOCKS, THREADS >> > (plaintext1, plaintext0, ciphertext1, ciphertext0, t0, t1, t2, t3, s8);
//    KLEIN64ExhaustiveSearch32CopiesS8SingleTable << <BLOCKS, THREADS >> > (plaintext1, plaintext0, ciphertext1, ciphertext0, t0, s8);
//    KLEIN64ExhaustiveSearch32CopiesS8SingleTableShift << <BLOCKS, THREADS >> > (plaintext1, plaintext0, ciphertext1, ciphertext0, t0, s8b, s8c);
//    KLEIN64ExhaustiveSearch32CopiesS8SingleTableShiftBytePerm << <BLOCKS, THREADS >> > (plaintext1, plaintext0, ciphertext1, ciphertext0, t0, s8b, s8c);
      KLEIN80ExhaustiveSearch32CopiesS8SingleTable << <BLOCKS, THREADS >> > (plaintext1, plaintext0, ciphertext5, ciphertext4, t0, s8);
//    KLEIN96ExhaustiveSearch32CopiesS8SingleTable << <BLOCKS, THREADS >> > (plaintext1, plaintext0, ciphertext3, ciphertext2, t0, s8);
//    KLEIN64ExhaustiveSearch32CopiesS8SingleTableS << <BLOCKS, THREADS >> > (plaintext1, plaintext0, ciphertext1, ciphertext0, t0, s8);
    hipDeviceSynchronize();
    printf("Time elapsed: %f sec\n", float(clock() - beginTime) / CLOCKS_PER_SEC);
    printf("-------------------------------\n");

    hipFree(t0);    hipFree(t1);    hipFree(t2);    hipFree(t3);    hipFree(s4); hipFree(s8); hipFree(s8b); hipFree(s8c);
}
void sboxshift() {
    for (int i = 0; i < 256; i++) printf("0x%x, ", S8[i] << 16); printf("\n");
}
int main(void) {
	hipSetDevice(0);
//	KLEIN64();   printf("\n\n");
//  KLEIN64_table_based();
//  calculate_tables();
    ExhaustiveSearch();
//     sboxshift();
//    generate_s8();
	printf("%s\n", hipGetErrorString(hipGetLastError()));
}

